#include "hip/hip_runtime.h"
#include "../include/cuda_transform.h"
#define LOOP_UNROLL


// Bilinear Interpolation
__device__ float bilinear(float q11, float q12, float q21, float q22, float scale)
{
    return (1.0f - scale)*(1.0f - scale)*q11 + (1.0f - scale)*scale*q12 + scale*(1.0f - scale)*q21 + scale*scale*q22;
}


//__global__ void __launch_bounds__(MAX_BLOCK_SIZE, MIN_BLOCKS_PER_SM) resize(uchar* d_input, size_t in_pitch, int height, int width, uchar* d_output, size_t out_pitch, float scale)
__global__ void resize(uchar* d_input, size_t in_pitch, int height, int width, uchar* d_output, size_t out_pitch, float scale)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    for (uint i = row; i < height; i += blockDim.y*gridDim.y)
        //#pragma unroll
        for (uint j = col; j < width; j += blockDim.x*gridDim.x)
        {
#ifdef LOOP_UNROLL
            if (threadIdx.y + 1 < blockDim.y)
            {
                int r = i*scale, c = j*scale;
                uchar *q11 = (uchar*)((char*)d_input + r*in_pitch) + c;
                uchar *q12 = (uchar*)((char*)d_input + (r + 1)*in_pitch) + c;
                uchar *q21 = (uchar*)((char*)d_input + r*in_pitch) + c + 1;
                uchar *q22 = (uchar*)((char*)d_input + (r + 1)*in_pitch) + c + 1;

                // Bilinear Interpolation
                float p = bilinear(*q11, *q12, *q21, *q22, scale);
                uchar *outputPixel = (uchar*)((char*)d_output + i*out_pitch) + j;
                *outputPixel = (uchar)p;

                r = r + 1;
                q11 = (uchar*)((char*)d_input + r*in_pitch) + c;
                q12 = (uchar*)((char*)d_input + (r + 1)*in_pitch) + c;
                q21 = (uchar*)((char*)d_input + r*in_pitch) + c + 1;
                q22 = (uchar*)((char*)d_input + (r + 1)*in_pitch) + c + 1;

                p = bilinear(*q11, *q12, *q21, *q22, scale);
                outputPixel = (uchar*)((char*)d_output + (i + 1)*out_pitch) + j;
                *outputPixel = (uchar)p;
            }
#else
#pragma unroll
            for (uint k = 0; k < 2; k++)
            {
                if (threadIdx.y + 1 < blockDim.y)
                {

                    int r = i*scale + k, c = j*scale;
                    uchar *q11 = (uchar*)((char*)d_input + r*in_pitch) + c;
                    uchar *q12 = (uchar*)((char*)d_input + (r + 1)*in_pitch) + c;
                    uchar *q21 = (uchar*)((char*)d_input + r*in_pitch) + c + 1;
                    uchar *q22 = (uchar*)((char*)d_input + (r + 1)*in_pitch) + c + 1;

                    uchar *outputPixel = (uchar*)((char*)d_output + (i + k)*out_pitch) + j;

                    float p = bilinear(*q11, *q12, *q21, *q22, scale);
                    *outputPixel = (uchar)p;
                }
            }
#endif // 
        }
}


__global__ void tranpose(uchar *d_input, int height, int width, uchar *d_output)
{
    int row = 32 * blockIdx.y + threadIdx.y;
    int col = 32 * blockIdx.x + threadIdx.x;

    __shared__ int smem[32][32 + 1];
    if (row < height&&col < width)
    {
        //#pragma unroll
        for (size_t i = 0; i < 32; i += 8)
        {
            smem[threadIdx.y + i][threadIdx.x] = d_input[(row + i)*width + col];
        }
        __syncthreads();

        row = blockIdx.x*32 + threadIdx.y;
        col = blockIdx.y*32 + threadIdx.x;

        //#pragma unroll
        for (size_t i = 0; i < 32; i += 8)
        {
            d_output[(height - row - i)*width + width - col] = smem[threadIdx.x][threadIdx.y + i];
            //d_output[(width - col - i)*width + height - row] = smem[threadIdx.x][threadIdx.y + i];
        }
    }
}


void cudaResize(const cv::Mat & input, cv::Mat & output, float scale)
{
    int newRow = int(input.rows * scale);
    int newCol = int(input.cols * scale);
    output = cv::Mat(cv::Size(newCol, newRow), CV_8U, cv::Scalar(0));
    scale = 1.0f / scale;

    // define block size and thread size
    dim3 block_size(THREAD_MULTIPLE, 6);
    dim3 grid_size(output.cols / (4 * block_size.x), output.rows / (4 * block_size.y)); // I divide the image into 16 grid to increase ILP level.

    hipStream_t stream; hipStreamCreate(&stream);

    size_t in_pitch, out_pitch;
    uchar *d_input, *d_output;
    hipMallocPitch(&d_input, &in_pitch, sizeof(uchar)*input.cols, input.rows);
    hipMemcpy2DAsync(d_input, in_pitch, input.data, sizeof(uchar)*input.cols, sizeof(uchar)*input.cols, input.rows, hipMemcpyHostToDevice, stream);
    hipMallocPitch(&d_output, &out_pitch, sizeof(uchar)*output.cols, output.rows);

    resize <<<grid_size, block_size, 0, stream >>>(d_input, in_pitch, output.rows, output.cols, d_output, out_pitch, scale);
    hipDeviceSynchronize();

    hipMemcpy2D(output.data, sizeof(uchar)*output.cols, d_output, out_pitch, sizeof(uchar)*output.cols, output.rows, hipMemcpyDeviceToHost);

    // resource releasing
    hipStreamDestroy(stream);
    hipFree(d_input); hipFree(d_output);
}
#include "hip/hip_runtime.h"
#include "../include/cuda_gamma_correction.h"


__device__ uchar4 gamma_handler(uchar4 pixels, float gamma)
{
    /*uchar p0 = __powf((float)pixels.x / 255.0f, gamma)*255.0f;
    uchar p1 = __powf((float)pixels.y / 255.0f, gamma)*255.0f;
    uchar p2 = __powf((float)pixels.z / 255.0f, gamma)*255.0f;
    uchar p3 = __powf((float)pixels.w / 255.0f, gamma)*255.0f;*/
    float t = fabsf(gamma - 1.0f);
    uchar p0 = __powf((float)pixels.x, gamma)*__powf(255.0f, t);
    uchar p1 = __powf((float)pixels.y, gamma)*__powf(255.0f, t);
    uchar p2 = __powf((float)pixels.z, gamma)*__powf(255.0f, t);
    uchar p3 = __powf((float)pixels.w, gamma)*__powf(255.0f, t);

    return make_uchar4(p0, p1, p2, p3);
}


template<int channels>
__global__ void gamma_correction(uchar *d_input, int height, int width, float gamma, uchar *d_output)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    for (int i = row; i < channels * height / 4; i += blockDim.y*gridDim.y) // stride by 4 byte
        for (int j = col; j < width; j += blockDim.x*gridDim.x)
        {
            uchar4 p = reinterpret_cast<uchar4*>(d_input)[i*width + j];
            reinterpret_cast<uchar4*>(d_output)[i*width + j] = gamma_handler(p, gamma);
        }
}


void cudaGammaCorrection(const cv::Mat & input, cv::Mat & output, float gamma)
{
    int channels = input.channels();

    output = cv::Mat(input.size(), input.type(), cv::Scalar(0));
    // define block size and
    dim3 block_size(THREAD_MULTIPLE, 6);
    // divide the image into 16 grids, smaller grid do more things, improve performance a lot.
    dim3 grid_size(input.cols / (4 * block_size.x), input.rows / (4 * block_size.y));

    uchar *d_input, *d_output;
    hipStream_t stream;
    CUDA_CALL(hipStreamCreate(&stream));
    CUDA_CALL(hipMalloc(&d_input, sizeof(uchar)*input.cols*input.rows*channels));
    CUDA_CALL(hipMemcpyAsync(d_input, input.data, sizeof(uchar)*input.cols*input.rows*channels, hipMemcpyHostToDevice, stream));
    CUDA_CALL(hipMalloc(&d_output, sizeof(uchar)*input.cols*input.rows*channels));

    switch (channels)
    {
    case 1:  gamma_correction<1> <<<grid_size, block_size, 0, stream>>>(d_input, input.rows, input.cols, gamma, d_output); break;
    case 3:  gamma_correction<3> <<<grid_size, block_size, 0, stream>>>(d_input, input.rows, input.cols, gamma, d_output); break;
    default: break;
    }
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(output.data, d_output, sizeof(uchar)*output.cols*output.rows*channels, hipMemcpyDeviceToHost));

    // resources releasing
    CUDA_CALL(hipStreamDestroy(stream));
    CUDA_CALL(hipFree(d_input)); CUDA_CALL(hipFree(d_output));
}
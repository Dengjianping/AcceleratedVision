#include "hip/hip_runtime.h"
#include "../include/cuda_histogram.h"

#define BINS 256


__global__ void histogram(uchar *d_input, int height, int width, uint *d_output)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = blockDim.x*blockIdx.x + threadIdx.x;

    for (int i = row; i < height; i += blockDim.y*gridDim.y)
        for (int j = col; j < width; j += blockDim.x*gridDim.x)
        {
            // atomic function at device with cc 2.1 has a really performance, cc > 3.5 has better performance
            // see this link, https://devblogs.nvidia.com/gpu-pro-tip-fast-histograms-using-shared-atomics-maxwell/
            atomicAdd(&d_output[d_input[i*width + j]], 1);
        }
}


void cudaHistogram(const cv::Mat & input, uint *hist)
{
    hist = new uint[BINS];
    // define block size and
    dim3 block_size(THREAD_MULTIPLE, 8);
    // divide the image into 16 grids, smaller grid do more things, improve performance a lot.
    dim3 grid_size(input.cols / (4 * block_size.x), input.rows / (4 * block_size.y));

    uchar *d_input; uint *d_output;
    hipStream_t stream;
    CUDA_CALL(hipStreamCreate(&stream));
    CUDA_CALL(hipMalloc(&d_input, sizeof(uchar)*input.cols*input.rows));
    CUDA_CALL(hipMemcpyAsync(d_input, input.data, sizeof(uchar)*input.cols*input.rows, hipMemcpyHostToDevice, stream));
    CUDA_CALL(hipMalloc(&d_output, sizeof(uint)*BINS));
    CUDA_CALL(hipMemset(d_output, 0, sizeof(uint)*BINS));

    // calling kernel
    histogram <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output);
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpy(hist, d_output, sizeof(uint)*BINS, hipMemcpyDeviceToHost));

    // resources releasing
    CUDA_CALL(hipStreamDestroy(stream));
    CUDA_CALL(hipFree(d_input)); CUDA_CALL(hipFree(d_output));
}
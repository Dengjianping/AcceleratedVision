#include "hip/hip_runtime.h"
#include "../include/cuda_dilation.h"


__constant__ int dilation_mask[21]; // only support a mask size 21 * 21


template<int RADIUS>
__global__ void dilation(uchar *d_input, int height, int width, uchar *d_output)
{
    int row = blockDim.y*blockIdx.y + threadIdx.y;
    int col = 4 * blockDim.x*blockIdx.x + threadIdx.x; // each thread handle 4 pixels

    static __shared__ int smem[6 + 2 * RADIUS][32 * 4 + 2 * RADIUS];
    for (int i = row; i < height; i += blockDim.y*gridDim.y)
    {
        int index = i*width + col;

        // separatable row computation
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS] = d_input[index];
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS + 32] = d_input[index + 32];
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS + 64] = d_input[index + 64];
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS + 96] = d_input[index + 96];

        // up and bottom row
        if (threadIdx.y < RADIUS)
        {
            int global_index = (i - RADIUS)*width + col;
            smem[threadIdx.y][threadIdx.x + RADIUS] = d_input[global_index];
            smem[threadIdx.y][threadIdx.x + RADIUS + 32] = d_input[global_index + 32];
            smem[threadIdx.y][threadIdx.x + RADIUS + 64] = d_input[global_index + 64];
            smem[threadIdx.y][threadIdx.x + RADIUS + 96] = d_input[global_index + 96];
        }
        if (threadIdx.y + RADIUS >= 6)
        {
            int global_index = (i + RADIUS)*width + col;
            smem[threadIdx.y + 2 * RADIUS][threadIdx.x + RADIUS] = d_input[global_index];
            smem[threadIdx.y + 2 * RADIUS][threadIdx.x + RADIUS + 32] = d_input[global_index + 32];
            smem[threadIdx.y + 2 * RADIUS][threadIdx.x + RADIUS + 64] = d_input[global_index + 64];
            smem[threadIdx.y + 2 * RADIUS][threadIdx.x + RADIUS + 96] = d_input[global_index + 96];
        }

        // left and right column
        if (threadIdx.x < RADIUS)
            smem[threadIdx.y + RADIUS][threadIdx.x] = d_input[i*width + (col - RADIUS)];
        if (threadIdx.x + RADIUS >= 32)
            smem[threadIdx.y + RADIUS][threadIdx.x + 2 * RADIUS + 96] = d_input[i*width + col + RADIUS + 96];
        __syncthreads();

        int sum = 0, sum_32 = 0, sum_64 = 0, sum_96 = 0;
        #pragma unroll
        for (int j = -RADIUS; j <= RADIUS; j++)
        {
            sum = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS - j][threadIdx.x + RADIUS]) | sum;
            sum_32 = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS - j][threadIdx.x + RADIUS + 32]) | sum_32;
            sum_64 = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS - j][threadIdx.x + RADIUS + 64]) | sum_64;
            sum_96 = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS - j][threadIdx.x + RADIUS + 96]) | sum_96;
        }
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS] = sum;
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS + 32] = sum_32;
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS + 64] = sum_64;
        smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS + 96] = sum_96;
        __syncthreads();

        sum = 0, sum_32 = 0, sum_64 = 0, sum_96 = 0;
        #pragma unroll
        for (int j = -RADIUS; j <= RADIUS; j++)
        {
            sum = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS - j]) | sum;
            sum_32 = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS - j + 32]) | sum_32;
            sum_64 = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS - j + 64]) | sum_64;
            sum_96 = (dilation_mask[RADIUS + j] | smem[threadIdx.y + RADIUS][threadIdx.x + RADIUS - j + 96]) | sum_96;
        }

        d_output[index] = sum;
        d_output[index + 32] = sum_32;
        d_output[index + 64] = sum_64;
        d_output[index + 96] = sum_96;
    }
}

void cudaDilation(const cv::Mat & input, int kernel_size, int dilation_times, cv::Mat & output)
{
    if (input.channels() != 1)return;
    output = cv::Mat(input.size(), input.type(), cv::Scalar(0));

    int diameter = 2 * kernel_size + 1;
    int *host_dilation_mask = new int[diameter];
    for (int i = 0; i < diameter; i++)host_dilation_mask[i] = 1;
    CUDA_CALL(hipMemcpyToSymbol(HIP_SYMBOL(dilation_mask), host_dilation_mask, sizeof(int)*diameter, 0, hipMemcpyHostToDevice));

    hipStream_t stream; CUDA_CALL(hipStreamCreate(&stream));

    uchar *d_input, *d_output;
    CUDA_CALL(hipMalloc(&d_input, sizeof(uchar)*input.rows*input.cols));
    CUDA_CALL(hipMemcpyAsync(d_input, input.data, sizeof(uchar)*input.rows*input.cols, hipMemcpyHostToDevice, stream));
    CUDA_CALL(hipMalloc(&d_output, sizeof(uchar)*input.rows*input.cols));

    // define block size and
    dim3 block_size(THREAD_MULTIPLE, 6);
    // divide the image into 16 grids, smaller grid do more things, improve performance a lot.
    dim3 grid_size(input.cols / (4 * block_size.x), input.rows / (4 * block_size.y));

    switch (kernel_size)
    {
    case 1: dilation<1> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 2: dilation<2> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 3: dilation<3> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 4: dilation<4> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 5: dilation<5> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 6: dilation<6> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 7: dilation<7> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 8: dilation<8> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 9: dilation<9> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    case 10: dilation<10> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output); break;
    default: break;
    }

    /*for (int i = 0; i < dilation_times; i++)
    {
        dilation<1> <<<grid_size, block_size, 0, stream>>> (d_input, input.rows, input.cols, d_output);
        hipMemset(d_input, 0, sizeof(uchar)*output.rows*output.cols);
        hipMemcpyAsync(d_input, d_output, sizeof(uchar)*input.rows*input.cols, hipMemcpyDeviceToDevice, stream);
        if (i != dilation_times - 1)
            hipMemset(d_output, 0, sizeof(uchar)*output.rows*output.cols);
    }*/
    CUDA_CALL(hipDeviceSynchronize());

    CUDA_CALL(hipMemcpyAsync(output.data, d_output, sizeof(uchar)*input.rows*input.cols, hipMemcpyDeviceToHost, stream));
    CUDA_CALL(hipFree(d_input)); CUDA_CALL(hipFree(d_output)); CUDA_CALL(hipStreamDestroy(stream));
    delete[] host_dilation_mask;
}